#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<math.h>
#include<stdlib.h>
#include<time.h>
#include <chrono>
#include <cstdlib>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
// to remove intellisense highlighting
#include <>

const int ntpb = 512;
using namespace std;
using namespace std::chrono;

void calculatePI(int n, float* h_a) {
	float x, y;
	int hit;
	srand(time(NULL));
	for (int j = 0; j < n; j++) {
		hit = 0;
		x = 0; 
		y = 0;
		for (int i = 0; i < n; i++) {
			x = float(rand()) / float(RAND_MAX);
			y = float(rand()) / float(RAND_MAX);
			if (y <= sqrt(1 - (x * x))) {
				hit += 1;
			}
		}

		h_a[j] = 4 * float(hit) / float(n);

	}
}

__global__ void setRng(hiprandState *rng) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(123456, idx, 0, &rng[idx]);
}


__global__ void calPI(float* d_a, int n, hiprandState *rng) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	unsigned int counter = 0;
	while (counter < n) {
		float x = hiprand_uniform(&rng[idx]);
		float y = hiprand_uniform(&rng[idx]);

		if (y <= sqrt(1 - (x * x))) {
			d_a[idx]++;
		}
		counter++;
	}
	d_a[idx] = 4.0 * (float(d_a[idx])) / float(n);
}



void reportTime(const char* msg, steady_clock::duration span) {
	auto ms = duration_cast<milliseconds>(span);
	std::cout << msg << " took - " <<
		ms.count() << " millisecs" << std::endl;
}
int main(int argc, char* argv[]) {

	if (argc != 2) {
		std::cerr << argv[0] << ": invalid number of arguments\n";
		std::cerr << "Usage: " << argv[0] << "  size_of_matrices\n";
		return 1;
	}
	int n = std::atoi(argv[1]); // scale
	int nblks = (n + ntpb - 1) / ntpb;
	cout << "scale: " << n << endl << endl;
	steady_clock::time_point ts, te;

	float* cpu_a;
	cpu_a = new float[n];

	ts = steady_clock::now();
	calculatePI(n, cpu_a);
	te = steady_clock::now();
	reportTime("CPU", te - ts);




	ofstream h_file;
	h_file.open("h_result.txt");
	float cpuSum = 0.0f;
	for (int i = 0; i < n; i++) {
		cpuSum += cpu_a[i];
		h_file << "Host: " << cpu_a[i] << endl;
	}
	cpuSum = cpuSum / (float)n;
	cout << "CPU Result: " << cpuSum << endl;
	h_file.close();

	cout << endl;
	////////////////////////////////////////

	hiprandState *d_rng;
	float* d_a;
	float* h_a;
	h_a = new float[n];

	hipMalloc((void**)&d_a, n * sizeof(float));
	hipMalloc((void**)&d_rng, n * sizeof(hiprandState));

	ts = steady_clock::now();

	setRng << < nblks, ntpb >> > (d_rng);
	hipDeviceSynchronize();	// synchronize [new added]
	calPI << <nblks, ntpb >> > (d_a, n, d_rng);
	hipDeviceSynchronize();

	te = steady_clock::now();
	reportTime("GPU", te - ts);

	hipMemcpy(h_a, d_a, n * sizeof(float), hipMemcpyDeviceToHost);


	ofstream d_file;
	d_file.open("d_result.txt");
	float gpuSum = 0.0f;
	for (int i = 0; i < n; i++) {
		gpuSum += h_a[i];
		d_file << "Device: " << h_a[i] << endl;
	}
	gpuSum = gpuSum / (float)n;
	cout << "GPU Result: " << gpuSum << endl;
	d_file.close();


	delete[] cpu_a;
	delete[] h_a;
	hipFree(d_a);
	hipFree(d_rng);

}


